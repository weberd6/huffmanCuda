#include "hip/hip_runtime.h"
#include "main.h"

__global__
void update_histo(unsigned int* d_in,
                unsigned int val1,
		unsigned int val2,
		unsigned int* d_indices,
                const size_t numElems)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  extern __shared__ int found[];

  found[0] = found[1] = false;
  __syncthreads();

  if ((i < numElems) && (0 == atomicCAS(&found[0], 0, 1)) && (d_in[i] == val1)) {
    d_in[i] = 0xFFFFFFFF;
    d_indices[0] = i;
    goto done;
  }

  if ((i < numElems) && (0 == atomicCAS(&found[1], 0, 1)) && (d_in[i] == val2)) {
    d_in[i] = val1 + val2;
    d_indices[1] = i;
  }

  done:
    return;
}

__global__
void reduce_min2(const unsigned int* d_in,
                unsigned int* d_vals,
                const size_t numElems)
{ 
 int i = threadIdx.x + blockDim.x * blockIdx.x;
 int threadId = threadIdx.x;

  extern __shared__ unsigned int shdata_min[];

  if (i >= numElems)
    shdata_min[threadId] = 0xFFFFFFFF;  // Infinity
  else
    shdata_min[threadId] = d_in[i];

  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 1; s >>= 1)
  {
    if (threadId < s)
    {
      shdata_min[threadId] = min(shdata_min[threadId], shdata_min[threadId + s]);
      shdata_min[threadId + 1] = min(shdata_min[threadId + 1], shdata_min[threadId + s + 1]);
    }
    __syncthreads();
  }

  if (threadId == 0)
  {
    d_vals[0] = shdata_min[0];
    d_vals[1] = shdata_min[1];
  }
}

void update_histo_and_get_min_indices(unsigned int* d_in,
				unsigned int val1,
				unsigned int val2,
				unsigned int* d_indices,
				const size_t numElems)
{
  update_histo<<<1, numElems, 2*sizeof(unsigned int)>>>(d_in, val1, val2, d_indices, numElems);
}

// Assumes numElems is a power of 2 and is less than 1024
void get_minimum2(const unsigned int* d_in,
		  const size_t numElems,
		  unsigned int* d_vals)
{
  reduce_min2<<<1, numElems, numElems*sizeof(int)>>>(d_in, d_vals, numElems);
}



