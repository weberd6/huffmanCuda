#include "hip/hip_runtime.h"
#include "main.h"
#include "node.h"

#include <ctime>
#include <string>
#include <cstring>
#include <sys/stat.h>
#include <fstream>
#include <cassert>

long getFileSize(std::string filename)
{
    struct stat stat_buf;
    int rc = stat(filename.c_str(), &stat_buf);
    return rc == 0 ? stat_buf.st_size : -1;
}

void print_frequencies(unsigned int* freq, const size_t size)
{
	for (unsigned int i = 0; i < size; i++)
		std::cout << freq[i] << std::endl;
}

void parallel_huffman(char* data, unsigned int num_bytes)
{
	const unsigned int NUM_VALS = 256;

	unsigned char* d_vals;
	unsigned int* d_frequencies;
	unsigned int* d_min_frequencies;
	unsigned int* d_min_indices;

	unsigned int* h_frequencies = (unsigned int*)malloc(NUM_VALS*sizeof(unsigned int));
	unsigned int* h_min_frequencies = (unsigned int*)malloc(2*sizeof(unsigned int));
	unsigned int* h_min_indices = (unsigned int*)malloc(2*sizeof(unsigned int));	

	hipMalloc(&d_vals, num_bytes*sizeof(unsigned char));
	hipMalloc(&d_frequencies, NUM_VALS*sizeof(unsigned int));
	hipMalloc(&d_min_frequencies, 2*sizeof(unsigned int));
	hipMalloc(&d_min_indices, 2*sizeof(unsigned int));

	hipMemcpy(d_vals, data, num_bytes*sizeof(unsigned char), hipMemcpyHostToDevice);

	computeHistogram(d_vals, d_frequencies, NUM_VALS, num_bytes);

	unsigned int count = NUM_VALS;
	unsigned int* d_count;
	hipMalloc(&d_count, sizeof(unsigned int));
	hipMemcpy(d_count, &count, sizeof(count), hipMemcpyHostToDevice);
	minimizeBins(d_frequencies, d_count, NUM_VALS);
	hipMemcpy(&count, d_count, sizeof(count), hipMemcpyDeviceToHost);

	hipMemcpy(h_frequencies, d_frequencies, NUM_VALS*sizeof(unsigned int), hipMemcpyDeviceToHost);

	unsigned int sum = 0;
	Node leaf_nodes[NUM_VALS];
	Node* node_by_index[NUM_VALS];

	for (int i = 0; i < NUM_VALS; i++) {
		if (h_frequencies[i] != 0xFFFFFFFF) sum += h_frequencies[i];
		leaf_nodes[i].frequency = h_frequencies[i];
		node_by_index[i] = &leaf_nodes[i];
	}

	Node* root;
	Node* l;
	Node* r;
	while (count > 1)
	{
		get_minimum2(d_frequencies, NUM_VALS, d_min_frequencies);
		hipMemcpy(h_min_frequencies, d_min_frequencies, 2*sizeof(unsigned int), hipMemcpyDeviceToHost);

//		std::cout << "Mins: " << h_min_frequencies[0] << " " << h_min_frequencies[1] << std::endl;

		update_histo_and_get_min_indices(d_frequencies, h_min_frequencies[0], h_min_frequencies[1], d_min_indices, NUM_VALS);
		hipMemcpy(h_min_indices, d_min_indices, 2*sizeof(unsigned int), hipMemcpyDeviceToHost);

		l = node_by_index[h_min_indices[0]];
		r = node_by_index[h_min_indices[1]];
//		std::cout << "Nodes: " << l->frequency << " " << r->frequency << std::endl;

		root = new Node(l, r, l->frequency + r->frequency);
		node_by_index[h_min_indices[1]] = root;

		count--;
	}

//	std::cout << "\nSize of file: " << num_bytes << " bytes" << std::endl;
//	std::cout << "Sum of frequencies: " << sum << std::endl;
//	std::cout << "Root huffman frequency: " <<  root->frequency << std::endl;
}

int main (int argc, char** argv) {
	
	bool run_parallel = true;
	std::string input_filename;

	if (1 == argc) {
		std::cout << "Missing argument: filename" << std::endl;
		exit(1);
	}

	for (int i = 1; i < argc; i++) {
		char* arg = argv[i];
		if (arg[0] == '-') {
			if (strcmp("-run_parallel", arg) == 0) {
				if (i == (argc-1)) {
					std::cout << "Missing argument for run_parallel" << std::endl;
					exit(1);
				}
				else {
					run_parallel = argv[i];
					i++;
				}
			}
		} else {
			input_filename = argv[i];
		}
	}

	if (input_filename.empty()) {
		std::cout << "Invalid arguments" << std::endl;
		exit(1);
	}

	long num_bytes = getFileSize(input_filename);
	char* data = new char[num_bytes];
	std::ifstream ifs(input_filename.c_str());
	if(!ifs) {
		std::cout << "Failed to open file: " << input_filename << std::endl;
	}

	ifs.read(data, num_bytes);

	std::clock_t start = std::clock();
	double duration;

	if (run_parallel) {
		parallel_huffman(data, num_bytes);
	} else {

	}

	duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;

	std::cout << "Elapsed time: " << duration*1000 << " ms" << std::endl;

	return 0;
}


