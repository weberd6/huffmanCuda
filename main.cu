#include "hip/hip_runtime.h"
#include "main.h"
#include "node.h"

#include <ctime>
#include <string>
#include <cstring>
#include <sys/stat.h>
#include <fstream>

void generate_code(Node *root, unsigned int code[], unsigned int length[]);

long getFileSize(std::string filename)
{
    struct stat stat_buf;
    int rc = stat(filename.c_str(), &stat_buf);
    return rc == 0 ? stat_buf.st_size : -1;
}

void print_frequencies(unsigned int* freq, const size_t size)
{
    for (unsigned int i = 0; i < size; i++)
        std::cout << freq[i] << std::endl;
}

void parallel_huffman(char* data, unsigned int num_bytes)
{
    const unsigned int NUM_VALS = 256;

    unsigned char* d_vals;
    unsigned int* d_frequencies;
    unsigned int* d_min_frequencies;
    unsigned int* d_min_indices;

    unsigned int* h_frequencies = (unsigned int*)malloc(NUM_VALS*sizeof(unsigned int));
    unsigned int* h_min_frequencies = (unsigned int*)malloc(2*sizeof(unsigned int));
    unsigned int* h_min_indices = (unsigned int*)malloc(2*sizeof(unsigned int));

    hipMalloc(&d_vals, num_bytes*sizeof(unsigned char));
    hipMalloc(&d_frequencies, NUM_VALS*sizeof(unsigned int));
    hipMalloc(&d_min_frequencies, 2*sizeof(unsigned int));
    hipMalloc(&d_min_indices, 2*sizeof(unsigned int));

    hipMemcpy(d_vals, data, num_bytes*sizeof(unsigned char), hipMemcpyHostToDevice);

    computeHistogram(d_vals, d_frequencies, NUM_VALS, num_bytes);

    unsigned int count = NUM_VALS;
    unsigned int* d_count;
    hipMalloc(&d_count, sizeof(unsigned int));
    hipMemcpy(d_count, &count, sizeof(count), hipMemcpyHostToDevice);
    minimizeBins(d_frequencies, d_count, NUM_VALS);
    hipMemcpy(&count, d_count, sizeof(count), hipMemcpyDeviceToHost);

    hipMemcpy(h_frequencies, d_frequencies, NUM_VALS*sizeof(unsigned int), hipMemcpyDeviceToHost);

    unsigned int sum = 0;
    Node leaf_nodes[NUM_VALS];
    Node* node_by_index[NUM_VALS];

    for (int i = 0; i < NUM_VALS; i++) {
        if (h_frequencies[i] != 0xFFFFFFFF) sum += h_frequencies[i];
        leaf_nodes[i].frequency = h_frequencies[i];
        leaf_nodes[i].symbol_index = i;
        node_by_index[i] = &leaf_nodes[i];
    }

    Node* root;
    Node* l;
    Node* r;
    while (count > 1)
    {
        get_minimum2(d_frequencies, NUM_VALS, d_min_frequencies);
        hipMemcpy(h_min_frequencies, d_min_frequencies, 2*sizeof(unsigned int), hipMemcpyDeviceToHost);

//      std::cout << "Mins: " << h_min_frequencies[0] << " " << h_min_frequencies[1] << std::endl;

        update_histo_and_get_min_indices(d_frequencies, h_min_frequencies[0], h_min_frequencies[1], d_min_indices, NUM_VALS);
        hipMemcpy(h_min_indices, d_min_indices, 2*sizeof(unsigned int), hipMemcpyDeviceToHost);

        l = node_by_index[h_min_indices[0]];
        r = node_by_index[h_min_indices[1]];
//      std::cout << "Nodes: " << l->frequency << " " << r->frequency << std::endl;

        root = new Node(l, r, l->frequency + r->frequency);
        node_by_index[h_min_indices[1]] = root;

        count--;
    }

//  std::cout << "\nSize of file: " << num_bytes << " bytes" << std::endl;
//  std::cout << "Sum of frequencies: " << sum << std::endl;
//  std::cout << "Root huffman frequency: " <<  root->frequency << std::endl;

    unsigned int codes[NUM_VALS];
    unsigned int lengths[NUM_VALS];

    memset(codes, 0, NUM_VALS*sizeof(unsigned int));
    memset(lengths, 0, NUM_VALS*sizeof(unsigned int));

    generate_code(root, codes, lengths);

//  for (unsigned int i = 0; i < NUM_VALS; i++) {
//      std::cout << i << ": " << codes[i] << "\t\t" << lengths[i] << std::endl;
//  }

    unsigned int* d_codes;
    unsigned int* d_lengths;
    unsigned int* d_data_lengths;
    unsigned int* d_lengths_partial_sums;
    unsigned char* d_encoded_data;
    hipMalloc(&d_codes, NUM_VALS*sizeof(unsigned int));
    hipMalloc(&d_lengths, NUM_VALS*sizeof(unsigned int));
    hipMalloc(&d_data_lengths, num_bytes*sizeof(unsigned int));
    hipMalloc(&d_lengths_partial_sums, num_bytes*sizeof(unsigned int));
    
    hipMemcpy(d_codes, codes, NUM_VALS*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_lengths, lengths, NUM_VALS*sizeof(unsigned int), hipMemcpyHostToDevice);

    compress_data(d_vals, d_codes, d_lengths, d_data_lengths, d_lengths_partial_sums, d_encoded_data, num_bytes);

    unsigned int* h_data_lengths = (unsigned int*)malloc(num_bytes*sizeof(unsigned int));
    unsigned int* h_lengths_partial_sums = (unsigned int*)malloc(num_bytes*sizeof(unsigned int));
    hipMemcpy(h_lengths_partial_sums, d_lengths_partial_sums, num_bytes*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(h_data_lengths, d_data_lengths, num_bytes*sizeof(unsigned int), hipMemcpyDeviceToHost);
    
//    for (unsigned int i = 0; i < num_bytes; i++) {
//        std::cout << i << ": " << h_data_lengths[i] << std::endl;
//    }
//    for (unsigned int i = 0; i < num_bytes; i++) {
 //       std::cout << i << ": " << h_lengths_partial_sums[i] << std::endl;
 //   }
    std::cout << "Compressed size: " << (h_data_lengths[num_bytes-1] + h_lengths_partial_sums[num_bytes-1])/8 << " bytes" << std::endl;

    free(h_frequencies);
    free(h_min_frequencies);
    free(h_min_indices);
    free(h_data_lengths);
    free(h_lengths_partial_sums);

    hipFree(d_vals);
    hipFree(d_frequencies);
    hipFree(d_min_frequencies);
    hipFree(d_min_indices);
    hipFree(d_count);
    hipFree(d_codes);
    hipFree(d_lengths);
    hipFree(d_data_lengths);
    hipFree(d_lengths_partial_sums);
    hipFree(d_encoded_data);
}

int main (int argc, char** argv) {

    bool run_parallel = true;
    std::string input_filename;

    if (1 == argc) {
        std::cout << "Missing argument: filename" << std::endl;
        exit(1);
    }

    for (int i = 1; i < argc; i++) {
        char* arg = argv[i];
        if (arg[0] == '-') {
            if (strcmp("-run_parallel", arg) == 0) {
                if (i == (argc-1)) {
                    std::cout << "Missing argument for run_parallel" << std::endl;
                    exit(1);
                }
                else {
                    run_parallel = atoi(argv[i]);
                    i++;
                }
            }
        } else {
            input_filename = argv[i];
        }
    }

    if (input_filename.empty()) {
        std::cout << "Invalid arguments" << std::endl;
        exit(1);
    }

    long num_bytes = getFileSize(input_filename);
    char* data = new char[num_bytes];
    std::ifstream ifs(input_filename.c_str());
    if(!ifs) {
        std::cout << "Failed to open file: " << input_filename << std::endl;
    }

    ifs.read(data, num_bytes);

    std::clock_t start = std::clock();
    double duration;

    if (run_parallel) {
        parallel_huffman(data, num_bytes);
    } else {

    }

    duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;

    std::cout << "Elapsed time: " << duration*1000 << " ms" << std::endl;

    return 0;
}


