#include "hip/hip_runtime.h"
#include <fstream>
#include <cstring>

#include "main.h"
#include "node.h"

void parallel_huffman_encode(unsigned char* data, unsigned int num_bytes, std::string filename)
{
    const unsigned int NUM_VALS = 256;

    unsigned char* d_vals;
    unsigned int* d_frequencies;
    unsigned int* d_min_frequencies;
    unsigned int* d_min_indices;

    unsigned int* h_frequencies = (unsigned int*)malloc(NUM_VALS*sizeof(unsigned int));
    unsigned int* h_min_frequencies = (unsigned int*)malloc(2*sizeof(unsigned int));
    unsigned int* h_min_indices = (unsigned int*)malloc(2*sizeof(unsigned int));

    hipMalloc(&d_vals, num_bytes*sizeof(unsigned char));
    hipMalloc(&d_frequencies, NUM_VALS*sizeof(unsigned int));
    hipMalloc(&d_min_frequencies, 2*sizeof(unsigned int));
    hipMalloc(&d_min_indices, 2*sizeof(unsigned int));

    hipMemcpy(d_vals, data, num_bytes*sizeof(unsigned char), hipMemcpyHostToDevice);

    hipMemset(d_frequencies, 0, NUM_VALS*sizeof(unsigned int));
    computeHistogram(d_vals, d_frequencies, NUM_VALS, num_bytes);

    unsigned int count = NUM_VALS;
    unsigned int* d_count;
    hipMalloc(&d_count, sizeof(unsigned int));
    hipMemcpy(d_count, &count, sizeof(count), hipMemcpyHostToDevice);
    minimizeBins(d_frequencies, d_count, NUM_VALS);
    hipMemcpy(&count, d_count, sizeof(count), hipMemcpyDeviceToHost);

    hipMemcpy(h_frequencies, d_frequencies, NUM_VALS*sizeof(unsigned int), hipMemcpyDeviceToHost);

    unsigned int sum = 0;
    Node leaf_nodes[NUM_VALS];
    Node* node_by_index[NUM_VALS];

    for (int i = 0; i < NUM_VALS; i++) {
        if (h_frequencies[i] != 0xFFFFFFFF) sum += h_frequencies[i];
        leaf_nodes[i].frequency = h_frequencies[i];
        leaf_nodes[i].symbol_index = i;
        node_by_index[i] = &leaf_nodes[i];
    }

    Node* root;
    Node* l;
    Node* r;
    while (count > 1) {
        get_minimum2(d_frequencies, NUM_VALS, d_min_frequencies);
        hipMemcpy(h_min_frequencies, d_min_frequencies, 2*sizeof(unsigned int), hipMemcpyDeviceToHost);

        update_histo_and_get_min_indices(d_frequencies, h_min_frequencies[0], h_min_frequencies[1], d_min_indices, NUM_VALS);
        hipMemcpy(h_min_indices, d_min_indices, 2*sizeof(unsigned int), hipMemcpyDeviceToHost);

        l = node_by_index[h_min_indices[0]];
        r = node_by_index[h_min_indices[1]];

        root = new Node(l, r, l->frequency + r->frequency);
        node_by_index[h_min_indices[1]] = root;

        count--;
    }

    unsigned int codes[NUM_VALS];
    unsigned int lengths[NUM_VALS];

    memset(codes, 0, NUM_VALS*sizeof(unsigned int));
    memset(lengths, 0, NUM_VALS*sizeof(unsigned int));

    generate_code(root, codes, lengths);

    unsigned int* d_codes;
    unsigned int* d_lengths;
    unsigned int* d_data_lengths;
    unsigned int* d_lengths_partial_sums;
    unsigned char* d_encoded_data;
    hipMalloc(&d_codes, NUM_VALS*sizeof(unsigned int));
    hipMalloc(&d_lengths, NUM_VALS*sizeof(unsigned int));
    hipMalloc(&d_data_lengths, num_bytes*sizeof(unsigned int));
    hipMalloc(&d_lengths_partial_sums, num_bytes*sizeof(unsigned int));
    
    hipMemcpy(d_codes, codes, NUM_VALS*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_lengths, lengths, NUM_VALS*sizeof(unsigned int), hipMemcpyHostToDevice);

    size_t compressed_num_bytes = get_compressed_length(d_lengths, d_vals, d_data_lengths, d_lengths_partial_sums, num_bytes);

    hipMalloc(&d_encoded_data, compressed_num_bytes*sizeof(unsigned char*));
    compress_data(d_vals, d_codes, d_lengths, d_lengths_partial_sums, d_encoded_data, compressed_num_bytes);

    unsigned char* h_encoded_data = (unsigned char*)malloc(compressed_num_bytes*sizeof(unsigned char));
    hipMemcpy(h_encoded_data, d_encoded_data, compressed_num_bytes*sizeof(unsigned char), hipMemcpyDeviceToHost);

    int lastindex = filename.find_last_of(".");
    std::string name = filename.substr(0, lastindex);
    std::string output_filename(name+".pc");
    std::ofstream ofs(output_filename.c_str(), std::ios::out | std::ios::trunc | std::ios::binary);

    serialize_tree(root, ofs);

    //TODO save length partial sum for every 32 lengths for block offset used to decode

    ofs.write(reinterpret_cast<const char*>(h_encoded_data), compressed_num_bytes);

    ofs.close();

    free(h_frequencies);
    free(h_min_frequencies);
    free(h_min_indices);
    free(h_encoded_data);

    hipFree(d_vals);
    hipFree(d_frequencies);
    hipFree(d_min_frequencies);
    hipFree(d_min_indices);
    hipFree(d_count);
    hipFree(d_codes);
    hipFree(d_lengths);
    hipFree(d_data_lengths);
    hipFree(d_lengths_partial_sums);
    hipFree(d_encoded_data);
}

void parallel_huffman_decode(std::ifstream& ifs, std::string filename)
{
    Node* h_root;
    deserialize_tree(h_root, ifs);

    unsigned int max_length = 0; // TODO Max reduce to find maximum length which will give the depth of the tree
    unsigned int array_size = 1 << (max_length+1);

    NodeArray* nodes = new NodeArray[array_size];
    std::memset(nodes, 0, array_size*sizeof(NodeArray));

    tree_to_array(nodes, 0, h_root);

    //TODO copy tree to device

    //TODO call decode kernel
}
